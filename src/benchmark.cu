#include "hip/hip_runtime.h"
#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <stdexcept>
#include <string>
#include <vector>

#include "engine/data.cuh"
#include "engine/verification.cuh"
#include "flsgpu/flsgpu-api.cuh"
#include "generated-bindings/kernel-bindings.cuh"

struct ProgramParameters {
  enums::DataType data_type;
  enums::Kernel kernel;
  std::string file;
  data::ValueRange<vbw_t> bit_width_range;
  data::ValueRange<uint16_t> ec_range;
  size_t n_values;
  enums::Print print_option;

  bool read_data_from_file() const { return file != "generate"; }
};

struct CLIArgs {
  std::string data_type;
  std::string kernel;
  std::string file;
  vbw_t start_vbw;
  vbw_t end_vbw;
  uint16_t start_ec;
  uint16_t end_ec;
  size_t n_vecs;
  uint32_t print_debug;

  CLIArgs(const int argc, char **argv) {
    constexpr int32_t CORRECT_ARG_COUNT = 10;
    if (argc != CORRECT_ARG_COUNT) {
      throw std::invalid_argument("Wrong arg count.\n");
    }

    int32_t argcounter = 0;
    data_type = argv[++argcounter];
    kernel = argv[++argcounter];
    file = argv[++argcounter];
    start_vbw = std::stoul(argv[++argcounter]);
    end_vbw = std::stoul(argv[++argcounter]);
    start_ec = std::stoul(argv[++argcounter]);
    end_ec = std::stoul(argv[++argcounter]);
    n_vecs = std::stoul(argv[++argcounter]);
    print_debug = std::stoul(argv[++argcounter]);
  }

  ProgramParameters parse() {
    return ProgramParameters{
        enums::string_to_data_type(data_type),
        enums::string_to_kernel(kernel),
        file,
        data::ValueRange<vbw_t>(start_vbw, end_vbw),
        data::ValueRange<uint16_t>(start_ec, end_ec),
        n_vecs * consts::VALUES_PER_VECTOR,
        static_cast<enums::Print>(print_debug),
    };
  }
};

template <typename T>
void execute_ffor_decompress(flsgpu::device::FFORColumn<T> column_device) {
  const unsigned unpack_n_values = 1;
  const std::vector<unsigned> unpack_n_vecs_set = {1, 4};

  for (const auto unpack_n_vecs : unpack_n_vecs_set) {
    for (size_t u{0};
         u < static_cast<size_t>(enums::Unpacker::StatefulBranchless); ++u) {
      const auto unpacker = static_cast<enums::Unpacker>(u);
      const T *out =
          bindings::decompress_column<T, flsgpu::device::FFORColumn<T>>(
              column_device, unpack_n_vecs, unpack_n_values, unpacker,
              enums::Patcher::None);
      delete[] out;
    }
  }
}
template <typename T>
int32_t execute_ffor_query(flsgpu::device::FFORColumn<T> column_device,
                           const bool column_contains_value,
                           const T value_to_query) {
  const unsigned unpack_n_values = 1;
  const std::vector<unsigned> unpack_n_vecs_set = {1, 4};
  int32_t failed = 0;

  for (const auto unpack_n_vecs : unpack_n_vecs_set) {
    for (size_t u{0};
         u < static_cast<size_t>(enums::Unpacker::StatefulBranchless); ++u) {
      const auto unpacker = static_cast<enums::Unpacker>(u);
      const bool answer =
          bindings::query_column<T, flsgpu::device::FFORColumn<T>>(
              column_device, unpack_n_vecs, unpack_n_values, unpacker,
              enums::Patcher::None, consts::as<T>::MAGIC_NUMBER);
      failed += answer;
    }
  }

  return failed;
}

template <typename T> int32_t execute_ffor(const ProgramParameters params) {
  using UINT_T = typename utils::same_width_uint<T>::type;
  auto results = std::vector<verification::ExecutionResult<T>>();

  int32_t failed = 0;

  for (vbw_t vbw{params.bit_width_range.min}; vbw <= params.bit_width_range.max;
       ++vbw) {
    if (params.kernel == enums::Kernel::Query) {
      auto [query_result, column] =
          data::columns::generate_binary_ffor_column<T>(
              params.n_values, data::ValueRange<vbw_t>(vbw),
              consts::MAX_UNPACK_N_VECS);
      auto column_device = column.copy_to_device();

      failed += execute_ffor_query(column_device, query_result,
                                   consts::as<T>::MAGIC_NUMBER);

      flsgpu::host::free_column(column_device);
      flsgpu::host::free_column(column);
    } else if (params.kernel == enums::Kernel::Decompress) {
      auto column = data::columns::generate_random_ffor_column<T>(
          params.n_values, data::ValueRange<vbw_t>(vbw),
          data::ValueRange<T>(0, 100), consts::MAX_UNPACK_N_VECS);
      auto column_device = column.copy_to_device();

      execute_ffor_decompress(column_device);

      flsgpu::host::free_column(column_device);
      flsgpu::host::free_column(column);
    }
  }

  return failed;
}

template <typename T> int32_t execute_alp(const ProgramParameters params) {
  return 0;
}

int main(int argc, char **argv) {
  CLIArgs args(argc, argv);
  ProgramParameters params = args.parse();

  int32_t exit_code = 0;
  bool print_debug = params.print_option != enums::Print::PrintNothing;
  switch (params.data_type) {
  case enums::DataType::U32:
    exit_code = execute_ffor<uint32_t>(params);

    break;
  case enums::DataType::U64:
    exit_code = execute_ffor<uint64_t>(params);
    break;
  case enums::DataType::F32:
    exit_code = execute_alp<float>(params);
    break;
  case enums::DataType::F64:
    exit_code = execute_alp<double>(params);
    break;
  }

	if (print_debug) {
		printf("Exit code: %d\n", exit_code);
	}

  if (params.print_option == enums::Print::PrintDebugExit0) {
    exit(0);
  }

  exit(exit_code);
}
