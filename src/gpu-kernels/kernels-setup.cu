#include <cstddef>
#include <cstdint>
#include <exception>
#include <stdexcept>

#include "../common/consts.hpp"
#include "alp.cuh"
#include "host-alp-utils.cuh"
#include "host-utils.cuh"
#include "kernels-bindings.hpp"
#include "kernels-global.cuh"

namespace kernels {
namespace fls {

#define FLS_DC(CASE, UNPACKER_T, N_VEC, N_VAL)                                 \
  case CASE: {                                                                 \
    kernels::device::fls::decompress_column<                                   \
        T, N_VEC, N_VAL, UNPACKER_T<T, N_VEC, N_VAL, BPFunctor<T>>>            \
        <<<n_blocks, n_threads>>>(device_out.get(), device_in.get(),           \
                                  value_bit_width);                            \
  } break;

template <typename T>
void verify_decompress_column(const KernelSpecification spec,
                              const T *__restrict in, T *__restrict out,
                              const size_t count,
                              const int32_t value_bit_width) {}

template <>
void verify_decompress_column(const KernelSpecification spec,
                              const uint32_t *__restrict in,
                              uint32_t *__restrict out, const size_t count,
                              const int32_t value_bit_width) {
  using T = uint32_t;
  const auto n_vecs = static_cast<uint32_t>(count / consts::VALUES_PER_VECTOR);
  const auto n_threads = utils::get_n_lanes<T>();
  const auto n_blocks = n_vecs / spec.n_vectors;
  const auto encoded_count =
      value_bit_width == 0
          ? 1
          : (count * static_cast<size_t>(value_bit_width)) / (8 * sizeof(T));

  GPUArray<T> device_in(encoded_count, in);
  GPUArray<T> device_out(count);

  switch (spec.spec) {
    FLS_DC(TEST_STATELESS_1_1, BitUnpackerStateless, 1, 1)
    FLS_DC(TEST_STATEFUL_1_1, BitUnpackerStateful, 1, 1)
    FLS_DC(TEST_STATELESS_BRANCHLESS_1_1, BitUnpackerStatelessBranchless, 1, 1)
    FLS_DC(TEST_STATEFUL_BRANCHLESS_1_1, BitUnpackerStatefulBranchless, 1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  device_out.copy_to_host(out);
}

#define FLS_QCCZ(CASE, UNPACKER_T, N_VEC, N_VAL)                               \
  case CASE: {                                                                 \
    kernels::device::fls::query_column_contains_zero<                          \
        T, N_VEC, N_VAL, UNPACKER_T<T, N_VEC, N_VAL, BPFunctor<T>>>            \
        <<<n_blocks, n_threads>>>(device_out.get(), device_in.get(),           \
                                  value_bit_width);                            \
  } break;

template <>
void query_column_contains_zero(const KernelSpecification spec,
                                const uint32_t *__restrict in,
                                uint32_t *__restrict out, const size_t count,
                                const int32_t value_bit_width) {
  using T = uint32_t;
  const auto n_vecs = static_cast<uint32_t>(count / consts::VALUES_PER_VECTOR);
  constexpr auto UNPACK_N_VECTORS = 1;
  const auto n_vectors_per_block = 2 * UNPACK_N_VECTORS;
  const auto n_blocks = n_vecs / n_vectors_per_block;
  const auto n_threads = utils::get_n_lanes<T>() * 2;

  const auto encoded_count =
      value_bit_width == 0
          ? 1
          : (count * static_cast<size_t>(value_bit_width)) / (8 * sizeof(T));

  GPUArray<T> device_in(encoded_count, in);
  GPUArray<T> device_out(1);

  switch (spec.spec) {
    FLS_QCCZ(QUERY_STATELESS_1_1, BitUnpackerStateless, 1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }

  CUDA_SAFE_CALL(hipDeviceSynchronize());

  device_out.copy_to_host(out);

  if (*out != 1) {
    *out = 0;
  }
}

template <typename T>
void query_column_contains_zero(const KernelSpecification spec,
                                const T *__restrict in, T *__restrict out,
                                const size_t count,
                                const int32_t value_bit_width) {}

} // namespace fls

namespace gpualp {

#define ALP_DC(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                      \
  case CASE: {                                                                 \
    device_column = transfer::copy_alp_column_to_gpu(data);                    \
    kernels::device::alp::decompress_column<                                   \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpColumn<T>>,                 \
        AlpColumn<T>><<<n_blocks, n_threads>>>(d_out.get(), device_column);    \
  } break;

#define ALP_DCE(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                     \
  case CASE: {                                                                 \
    device_extended_column = transfer::copy_alp_extended_column_to_gpu(data);  \
    kernels::device::alp::decompress_column<                                   \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpExtendedColumn<T>>,         \
        AlpExtendedColumn<T>>                                                  \
        <<<n_blocks, n_threads>>>(d_out.get(), device_extended_column);        \
  } break;

template <typename T>
void verify_decompress_column(const KernelSpecification spec, T *__restrict out,
                              const alp::AlpCompressionData<T> *data) {
  const auto count = data->size;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_blocks = n_vecs / 1;
  const auto n_threads = utils::get_n_lanes<T>();

  GPUArray<T> d_out(count);
  constant_memory::load_alp_constants<T>();

  AlpColumn<T> device_column;
  AlpExtendedColumn<T> device_extended_column;

  switch (spec.spec) {
    ALP_DC(TEST_STATELESS_1_1, BitUnpackerStateless,
           StatelessALPExceptionPatcher, 1, 1)
    ALP_DCE(TEST_STATEFUL_1_1, BitUnpackerStateless,
            PrefetchAllALPExceptionPatcher, 1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }

  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

  transfer::destroy_alp_column(device_column);
  transfer::destroy_alp_column(device_extended_column);
}

#define ALP_QCCM(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                    \
  case CASE: {                                                                 \
    device_column = transfer::copy_alp_column_to_gpu(data);                    \
    kernels::device::alp::query_column_contains_magic<                         \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpColumn<T>>,                 \
        AlpColumn<T>>                                                          \
        <<<n_blocks, n_threads>>>(d_out.get(), device_column, magic_value);    \
  } break;

#define ALP_QCCME(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                   \
  case CASE: {                                                                 \
    device_extended_column = transfer::copy_alp_extended_column_to_gpu(data);  \
    kernels::device::alp::query_column_contains_magic<                         \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpExtendedColumn<T>>,         \
        AlpExtendedColumn<T>><<<n_blocks, n_threads>>>(                        \
        d_out.get(), device_extended_column, magic_value);                     \
  } break;

template <typename T>
void query_column_contains_magic(const KernelSpecification spec,
                                 T *__restrict out,
                                 const alp::AlpCompressionData<T> *data,
                                 const T magic_value) {
  const auto count = data->size;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_warps_per_block = 2;
  const auto n_blocks = n_vecs / n_warps_per_block;
  const auto n_threads = n_warps_per_block * consts::THREADS_PER_WARP;

  GPUArray<T> d_out(1);
  constant_memory::load_alp_constants<T>();

  AlpColumn<T> device_column;
  AlpExtendedColumn<T> device_extended_column;

  switch (spec.spec) {
    ALP_QCCM(QUERY_STATELESS_1_1, BitUnpackerStateless,
             StatelessALPExceptionPatcher, 1, 1)
    ALP_QCCME(TEST_STATEFUL_1_1, BitUnpackerStateless,
              PrefetchAllALPExceptionPatcher, 1, 1)
    ALP_QCCM(TEST_STATELESS_1_1, kernels::device::fls::Baseline,
             kernels::device::alp::DummyALPExceptionPatcher, 1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }

  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

  if (*out != static_cast<T>(true)) {
    *out = static_cast<T>(false);
  }

  transfer::destroy_alp_column(device_column);
  transfer::destroy_alp_column(device_extended_column);
}
} // namespace gpualp

} // namespace kernels

template void kernels::fls::verify_decompress_column<uint8_t>(
    const kernels::KernelSpecification spec, const uint8_t *__restrict in,
    uint8_t *__restrict out, const size_t count, const int32_t value_bit_width);
template void kernels::fls::verify_decompress_column<uint16_t>(
    const kernels::KernelSpecification spec, const uint16_t *__restrict in,
    uint16_t *__restrict out, const size_t count,
    const int32_t value_bit_width);
template void kernels::fls::verify_decompress_column<uint64_t>(
    const kernels::KernelSpecification spec, const uint64_t *__restrict in,
    uint64_t *__restrict out, const size_t count,
    const int32_t value_bit_width);

template void kernels::fls::query_column_contains_zero<uint8_t>(
    const kernels::KernelSpecification spec, const uint8_t *__restrict in,
    uint8_t *__restrict out, const size_t count, const int32_t value_bit_width);
template void kernels::fls::query_column_contains_zero<uint16_t>(
    const kernels::KernelSpecification spec, const uint16_t *__restrict in,
    uint16_t *__restrict out, const size_t count,
    const int32_t value_bit_width);
template void kernels::fls::query_column_contains_zero<uint64_t>(
    const kernels::KernelSpecification spec, const uint64_t *__restrict in,
    uint64_t *__restrict out, const size_t count,
    const int32_t value_bit_width);

template void kernels::gpualp::verify_decompress_column<float>(
    const kernels::KernelSpecification spec, float *__restrict out,
    const alp::AlpCompressionData<float> *data);
template void kernels::gpualp::verify_decompress_column<double>(
    const kernels::KernelSpecification spec, double *__restrict out,
    const alp::AlpCompressionData<double> *data);

template void kernels::gpualp::query_column_contains_magic<float>(
    const ::kernels::KernelSpecification spec, float *__restrict out,
    const alp::AlpCompressionData<float> *data, const float magic_value);
template void kernels::gpualp::query_column_contains_magic<double>(
    const ::kernels::KernelSpecification spec, double *__restrict out,
    const alp::AlpCompressionData<double> *data, const double magic_value);
