#include <cstddef>
#include <cstdint>

#include "../common/consts.hpp"
#include "../common/utils.hpp"
#include "../gpu-common/gpu-utils.cuh"
#include "alp-benchmark-kernels-bindings.hpp"
#include "alp-benchmark-kernels-global.cuh"
#include "alp.cuh"
#include "src/alp/config.hpp"

namespace alp {
namespace gpu {
namespace bench {

template <typename T>
void decode_baseline(T *__restrict out, const T *in, const size_t count) {
  GPUArray<T> d_in(count, in);
  GPUArray<T> d_out(1);

  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_warps_per_block = 2;
  const auto n_blocks = n_vecs / n_warps_per_block;
  const auto n_threads = n_warps_per_block * consts::THREADS_PER_WARP;

  kernels::global::bench::decode_baseline<T, T, 1, 1>
      <<<n_blocks, n_threads>>>(d_out.get(), d_in.get());
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

	if (*out != static_cast<T>(true)) {
		*out = static_cast<T>(false);
	}
}

template <typename T>
void decode_complete_alp_vector(T *__restrict out,
                                const alp::AlpCompressionData<T> *data) {
  using UINT_T = typename utils::same_width_uint<T>::type;

  const auto count = data->size;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_warps_per_block = 2;
  const auto n_blocks = n_vecs / n_warps_per_block;
  const auto n_threads = n_warps_per_block * consts::THREADS_PER_WARP;

  GPUArray<T> d_out(1);
  GPUArray<UINT_T> d_ffor_array(count, data->ffor.array);

  GPUArray<UINT_T> d_ffor_bases(n_vecs, data->ffor.bases);
  GPUArray<uint8_t> d_bit_widths(n_vecs, data->ffor.bit_widths);
  GPUArray<uint8_t> d_exponents(n_vecs, data->exponents);
  GPUArray<uint8_t> d_factors(n_vecs, data->factors);

  GPUArray<T> d_exceptions(count, data->exceptions.exceptions);
  GPUArray<uint16_t> d_exception_positions(count, data->exceptions.positions);
  GPUArray<uint16_t> d_exception_counts(n_vecs, data->exceptions.counts);

  AlpColumn<T> alp_data = {
      d_ffor_array.get(),          d_ffor_bases.get(),      d_bit_widths.get(),
      d_exponents.get(),           d_factors.get(),         d_exceptions.get(),
      d_exception_positions.get(), d_exception_counts.get()};
  constant_memory::load_alp_constants<T>();

  kernels::global::bench::decode_complete_alp_vector<
      T, UINT_T, 1, 4>
      <<<n_blocks, n_threads>>>(d_out.get(), alp_data);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

	if (*out != static_cast<T>(true)) {
		*out = static_cast<T>(false);
	}
}

template <typename T>
void decode_complete_alprd_vector(T *__restrict out,
                                  const alp::AlpRdCompressionData<T> *data) {
  using UINT_T = typename utils::same_width_uint<T>::type;

  const auto count = data->size;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_blocks = n_vecs;

  GPUArray<T> d_out(1);

  GPUArray<uint16_t> d_left_ffor_array(count, data->left_ffor.array);
  GPUArray<uint16_t> d_left_ffor_bases(n_vecs, data->left_ffor.bases);
  GPUArray<uint8_t> d_left_bit_widths(n_vecs, data->left_ffor.bit_widths);

  GPUArray<UINT_T> d_right_ffor_array(count, data->right_ffor.array);
  GPUArray<UINT_T> d_right_ffor_bases(n_vecs, data->right_ffor.bases);
  GPUArray<uint8_t> d_right_bit_widths(n_vecs, data->right_ffor.bit_widths);

  GPUArray<uint16_t> d_left_parts_dicts(
      n_vecs * alp::config::MAX_RD_DICTIONARY_SIZE, data->left_parts_dicts);

  GPUArray<uint16_t> d_exceptions(count, data->exceptions.exceptions);
  GPUArray<uint16_t> d_exception_positions(count, data->exceptions.positions);
  GPUArray<uint16_t> d_exception_counts(n_vecs, data->exceptions.counts);

  AlpRdColumn<T> alp_data = {
      d_left_ffor_array.get(),     d_left_ffor_bases.get(),
      d_left_bit_widths.get(),     d_right_ffor_array.get(),
      d_right_ffor_bases.get(),    d_right_bit_widths.get(),
      d_left_parts_dicts.get(),    d_exceptions.get(),
      d_exception_positions.get(), d_exception_counts.get(),
  };
  constant_memory::load_alp_constants<T>();

  kernels::global::bench::decode_complete_alprd_vector<
      T, UINT_T, 1, utils::get_values_per_lane<T>()>
      <<<n_blocks, utils::get_n_lanes<T>()>>>(d_out.get(), alp_data);
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

	if (*out != static_cast<T>(true)) {
		*out = static_cast<T>(false);
	}
}

} // namespace bench
} // namespace gpu
} // namespace alp

template void alp::gpu::bench::decode_baseline<float>(float *__restrict out,
                                                      const float *in,
                                                      const size_t count);
template void alp::gpu::bench::decode_baseline<double>(double *__restrict out,
                                                       const double *in,
                                                       const size_t count);
template void alp::gpu::bench::decode_complete_alp_vector<float>(
    float *__restrict out, const alp::AlpCompressionData<float> *data);
template void alp::gpu::bench::decode_complete_alp_vector<double>(
    double *__restrict out, const alp::AlpCompressionData<double> *data);
template void alp::gpu::bench::decode_complete_alprd_vector<float>(
    float *__restrict out, const alp::AlpRdCompressionData<float> *data);
template void alp::gpu::bench::decode_complete_alprd_vector<double>(
    double *__restrict out, const alp::AlpRdCompressionData<double> *data);
