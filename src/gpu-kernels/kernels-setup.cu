#include <cstddef>
#include <cstdint>
#include <exception>
#include <stdexcept>

#include "../alp/alp-bindings.hpp"
#include "../common/consts.hpp"
#include "../common/runspec.hpp"
#include "alp.cuh"
#include "host-alp-utils.cuh"
#include "host-utils.cuh"
#include "kernels-global.cuh"

namespace kernels {
namespace fls {

#define FLS_DC(CASE, UNPACKER_T, N_VEC, N_VAL)                                 \
  case CASE: {                                                                 \
    kernels::device::fls::decompress_column<                                   \
        T, N_VEC, N_VAL, UNPACKER_T<T, N_VEC, N_VAL, BPFunctor<T>>>            \
        <<<n_blocks, n_threads>>>(device_out.get(), device_in.get(),           \
                                  value_bit_width);                            \
  } break;

template <typename T>
void verify_decompress_column(const runspec::KernelSpecification spec,
                              const T *__restrict in, T *__restrict out,
                              const size_t count,
                              const int32_t value_bit_width) {}

template <>
void verify_decompress_column<uint32_t>(const runspec::KernelSpecification spec,
                                        const uint32_t *__restrict in,
                                        uint32_t *__restrict out,
                                        const size_t count,
                                        const int32_t value_bit_width) {
  using T = uint32_t;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_threads = utils::get_n_lanes<T>();
  const auto n_blocks = n_vecs / spec.n_vectors;
  const auto encoded_count =
      value_bit_width == 0
          ? 1
          : (count * static_cast<size_t>(value_bit_width)) / (8 * sizeof(T));

  // The branchless version always does 1 access too many for each lane
  // That is why we allocate a little extra memory
  const size_t branchless_extra_access_buffer =
      sizeof(T) * utils::get_n_lanes<T>();
  GPUArray<T> device_in(encoded_count + branchless_extra_access_buffer, in);
  GPUArray<T> device_out(count);

  switch (spec.kernel) {
    FLS_DC(runspec::KernelOption::STATELESS_1_1, BitUnpackerStateless, 1, 1)
    FLS_DC(runspec::KernelOption::STATELESS_4_1, BitUnpackerStateless, 4, 1)
    FLS_DC(runspec::KernelOption::STATEFUL_1_1, BitUnpackerStateful, 1, 1)
    FLS_DC(runspec::KernelOption::STATEFUL_4_1, BitUnpackerStateful, 4, 1)
    FLS_DC(runspec::KernelOption::STATELESS_BRANCHLESS_1_1, BitUnpackerStatelessBranchless, 1, 1)
    FLS_DC(runspec::KernelOption::STATELESS_BRANCHLESS_4_1, BitUnpackerStatelessBranchless, 4, 1)
    FLS_DC(runspec::KernelOption::STATEFUL_BRANCHLESS_1_1, BitUnpackerStatefulBranchless, 1, 1)
    FLS_DC(runspec::KernelOption::STATEFUL_BRANCHLESS_4_1, BitUnpackerStatefulBranchless, 4, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  device_out.copy_to_host(out);
}

#define FLS_QCCZ(CASE, UNPACKER_T, N_VEC, N_VAL)                               \
  case CASE: {                                                                 \
    kernels::device::fls::query_column_contains_zero<                          \
        T, N_VEC, N_VAL, UNPACKER_T<T, N_VEC, N_VAL, BPFunctor<T>>>            \
        <<<n_blocks, n_threads>>>(device_out.get(), device_in.get(),           \
                                  value_bit_width);                            \
  } break;

template <typename T>
void query_column_contains_zero(const runspec::KernelSpecification spec,
                                const T *__restrict in, T *__restrict out,
                                const size_t count,
                                const int32_t value_bit_width) {}

template <>
void query_column_contains_zero<uint32_t>(
    const runspec::KernelSpecification spec, const uint32_t *__restrict in,
    uint32_t *__restrict out, const size_t count,
    const int32_t value_bit_width) {
  using T = uint32_t;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_threads = utils::get_n_lanes<T>();
  const auto n_blocks = n_vecs / spec.n_vectors;

  const auto encoded_count =
      value_bit_width == 0
          ? 1
          : (count * static_cast<size_t>(value_bit_width)) / (8 * sizeof(T));

  GPUArray<T> device_in(encoded_count, in);
  GPUArray<T> device_out(1);

  switch (spec.kernel) {
    FLS_QCCZ(runspec::KernelOption::STATELESS_1_1, BitUnpackerStateless,
             1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }

  CUDA_SAFE_CALL(hipDeviceSynchronize());

  device_out.copy_to_host(out);

  if (*out != 1) {
    *out = 0;
  }
}

} // namespace fls

namespace gpualp {

#define ALP_DC(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                      \
  case CASE: {                                                                 \
    device_column = transfer::copy_alp_column_to_gpu(data);                    \
    kernels::device::alp::decompress_column<                                   \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpColumn<T>>,                 \
        AlpColumn<T>><<<n_blocks, n_threads>>>(d_out.get(), device_column);    \
  } break;

#define ALP_DCE(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                     \
  case CASE: {                                                                 \
    device_extended_column = transfer::copy_alp_extended_column_to_gpu(data);  \
    kernels::device::alp::decompress_column<                                   \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpExtendedColumn<T>>,         \
        AlpExtendedColumn<T>>                                                  \
        <<<n_blocks, n_threads>>>(d_out.get(), device_extended_column);        \
  } break;

template <typename T>
void verify_decompress_column(const runspec::KernelSpecification spec,
                              T *__restrict out,
                              const alp::AlpCompressionData<T> *data) {
  const auto count = data->size;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_threads = utils::get_n_lanes<T>();
  const auto n_blocks = n_vecs / spec.n_vectors;

  GPUArray<T> d_out(count);
  constant_memory::load_alp_constants<T>();

  AlpColumn<T> device_column;
  AlpExtendedColumn<T> device_extended_column;

  switch (spec.kernel) {
    ALP_DC(runspec::KernelOption::STATELESS_1_1, BitUnpackerStateless,
           StatelessALPExceptionPatcher, 1, 1)
    ALP_DCE(runspec::KernelOption::STATEFUL_1_1, BitUnpackerStateless,
            PrefetchAllALPExceptionPatcher, 1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }

  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

  transfer::destroy_alp_column(device_column);
  transfer::destroy_alp_column(device_extended_column);
}

#define ALP_QCCM(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                    \
  case CASE: {                                                                 \
    device_column = transfer::copy_alp_column_to_gpu(data);                    \
    kernels::device::alp::query_column_contains_magic<                         \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpColumn<T>>,                 \
        AlpColumn<T>>                                                          \
        <<<n_blocks, n_threads>>>(d_out.get(), device_column, magic_value);    \
  } break;

#define ALP_QCCME(CASE, UNPACKER_T, PATCHER_T, N_VEC, N_VAL)                   \
  case CASE: {                                                                 \
    device_extended_column = transfer::copy_alp_extended_column_to_gpu(data);  \
    kernels::device::alp::query_column_contains_magic<                         \
        T, N_VEC, N_VAL,                                                       \
        AlpUnpacker<T, N_VEC, N_VAL,                                           \
                    UNPACKER_T<T, N_VEC, N_VAL, ALPFunctor<T>>,                \
                    PATCHER_T<T, N_VEC, N_VAL>, AlpExtendedColumn<T>>,         \
        AlpExtendedColumn<T>><<<n_blocks, n_threads>>>(                        \
        d_out.get(), device_extended_column, magic_value);                     \
  } break;

template <typename T>
void query_column_contains_magic(const runspec::KernelSpecification spec,
                                 T *__restrict out,
                                 const alp::AlpCompressionData<T> *data,
                                 const T magic_value) {
  const auto count = data->size;
  const auto n_vecs = utils::get_n_vecs_from_size(count);
  const auto n_threads = utils::get_n_lanes<T>();
  const auto n_blocks = n_vecs / spec.n_vectors;

  GPUArray<T> d_out(1);
  constant_memory::load_alp_constants<T>();

  AlpColumn<T> device_column;
  AlpExtendedColumn<T> device_extended_column;

  switch (spec.kernel) {
    ALP_QCCM(runspec::KernelOption::STATELESS_1_1, BitUnpackerStateless,
             StatelessALPExceptionPatcher, 1, 1)
    ALP_QCCME(runspec::KernelOption::STATEFUL_1_1, BitUnpackerStateless,
              PrefetchAllALPExceptionPatcher, 1, 1)
  default: {
    throw std::invalid_argument("Did not find this spec");
  } break;
  }

  CUDA_SAFE_CALL(hipDeviceSynchronize());

  d_out.copy_to_host(out);

  if (*out != static_cast<T>(true)) {
    *out = static_cast<T>(false);
  }

  transfer::destroy_alp_column(device_column);
  transfer::destroy_alp_column(device_extended_column);
}
} // namespace gpualp

} // namespace kernels

template void kernels::fls::verify_decompress_column<uint8_t>(
    const runspec::KernelSpecification spec, const uint8_t *__restrict in,
    uint8_t *__restrict out, const size_t count, const int32_t value_bit_width);
template void kernels::fls::verify_decompress_column<uint16_t>(
    const runspec::KernelSpecification spec, const uint16_t *__restrict in,
    uint16_t *__restrict out, const size_t count,
    const int32_t value_bit_width);
template void kernels::fls::verify_decompress_column<uint64_t>(
    const runspec::KernelSpecification spec, const uint64_t *__restrict in,
    uint64_t *__restrict out, const size_t count,
    const int32_t value_bit_width);

template void kernels::fls::query_column_contains_zero<uint8_t>(
    const runspec::KernelSpecification spec, const uint8_t *__restrict in,
    uint8_t *__restrict out, const size_t count, const int32_t value_bit_width);
template void kernels::fls::query_column_contains_zero<uint16_t>(
    const runspec::KernelSpecification spec, const uint16_t *__restrict in,
    uint16_t *__restrict out, const size_t count,
    const int32_t value_bit_width);
template void kernels::fls::query_column_contains_zero<uint64_t>(
    const runspec::KernelSpecification spec, const uint64_t *__restrict in,
    uint64_t *__restrict out, const size_t count,
    const int32_t value_bit_width);

template void kernels::gpualp::verify_decompress_column<float>(
    const runspec::KernelSpecification spec, float *__restrict out,
    const alp::AlpCompressionData<float> *data);
template void kernels::gpualp::verify_decompress_column<double>(
    const runspec::KernelSpecification spec, double *__restrict out,
    const alp::AlpCompressionData<double> *data);

template void kernels::gpualp::query_column_contains_magic<float>(
    const ::runspec::KernelSpecification spec, float *__restrict out,
    const alp::AlpCompressionData<float> *data, const float magic_value);
template void kernels::gpualp::query_column_contains_magic<double>(
    const ::runspec::KernelSpecification spec, double *__restrict out,
    const alp::AlpCompressionData<double> *data, const double magic_value);
